// #include <stdio.h>
// #include <assert.h>
// #include <fstream>
// #include <sstream>
// #include <iostream>
// #include <cmath>
// #include <sys/stat.h>
// #include <cmath>
// #include <time.h>
// #include <cuda_runtime_api.h>
// #include <cublas_v2.h>
// #include <memory>
// #include <string.h>
// #include <cstdint>
 
// #define M 4
// #define N 1
// #define K 3
 
// void printMatrix(float (*matrix)[N], int row, int col) {
//     for(int i=0;i<row;i++)
//     {
//         std::cout << std::endl;
//         std::cout << " [ ";
//         for (int j=0; j<col; j++) {
//          std::cout << matrix[i][j] << " ";
//         }
//         std::cout << " ] ";
//     }
//     std::cout << std::endl;
// }
 
// int main(void)
// {   
//     // https://claude.ai/chat/487387e8-c06e-42f9-b946-cf6a4846652a
//     float alpha=1.0;
//     float beta=1.0;
//     float h_param[M][K]={ {1,2,3},{1,2,3},{1,2,3},{1,2,3} };
//     float h_obj[K][N]={ {-4}, {-3}, {-5}};
//     float h_obj_constant[M][N] = {0};
//     // float h_obj_constant[4] = {0};
//     // float h_param[K]={ 1,2,3 };
//     // float h_obj[K]={ -4 -3 -5};
//     // float h_obj_constant[M][N] = {0};
//     float *d_param,*d_obj,*d_obj_constant;
//     cudaMalloc((void**)&d_param,M*K*sizeof(float));
//     cudaMalloc((void**)&d_obj,K*N*sizeof(float));
//     cudaMalloc((void**)&d_obj_constant,M*N*sizeof(float));
//     cudaMemcpy(d_param,&h_param,M*K*sizeof(float),cudaMemcpyHostToDevice);
//     cudaMemcpy(d_obj,&h_obj,K*N*sizeof(float),cudaMemcpyHostToDevice);
//     cudaMemcpy(d_obj_constant,&h_obj_constant,M*N*sizeof(float), cudaMemcpyHostToDevice);
//     cublasHandle_t handle;
//     cublasCreate(&handle);

//     printf("CHECK OBJ CONSTANT MATRIX\n");
//     cudaMemcpy(h_obj_constant, d_obj_constant, M * N * sizeof(float), cudaMemcpyDeviceToHost);
//     printMatrix(h_obj_constant, M, N);
//     // for(int i = 0; i < M; ++i){
//     //     for(int j = 0; j < N; ++j){
//     //         // printf("row:%d col:%d objective mat:%f ", i, j, h_objective_matrix[i * col_obj +j]);
//     //         printf("obj_constant_matrix[%d,%d]=%f ", i, j, h_obj_constant[i * N + j]);
//     //     }
//     //     printf("\n");
//     // }

//     printf("%d %d %d %d %d %d\n", M, N, K, K,N,M);
//     // cublasSgemm(handle,CUBLAS_OP_T,CUBLAS_OP_T, M, N, K,&alpha,d_obj, N, d_param, K,&beta, d_obj_constant, N);
//     cublasSgemm(handle, CUBLAS_OP_T, CUBLAS_OP_T, 
//     M,    // M=64 
//     N,    // N=1
//     K,    // K=3 
//     &alpha,
//     d_param, K,   // A的leading dimension=3
//     d_obj, N,     // B的leading dimension=1  
//     &beta,
//     d_obj_constant, M); // C的leading dimension=64
//     // cublasSgemm(handle, CUBLAS_OP_N, CUBLAS_OP_N, 
//     // N,    // C的列数 = 1
//     // M,    // C的行数 = 64
//     // K,    // A的列数 = 3
//     // &alpha,
//     // d_obj, N,        // B的主维度 = 3
//     // d_param, K,      // A的主维度 = 64
//     // &beta, 
//     // d_obj_constant, N); // C的主维度 = 1
//     cudaMemcpy(h_obj_constant,d_obj_constant,M*N*sizeof(float),cudaMemcpyDeviceToHost);//此处的h_C是按列存储的CT
//     printMatrix(h_obj_constant, M, N);//按行读取h_C相当于做了CTT=C的结果
//     return 0;
// }



#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>
#include <fstream>
#include <sstream>
#include <iostream>
#include <cmath>
#include <sys/stat.h>
#include <cmath>
#include <time.h>
#include <hip/hip_runtime_api.h>
#include <hipblas.h>
#include <memory>
#include <string.h>
#include <cstdint>
 
#define M 2
#define N 2
#define K 4


void printMatrix2(float* matrix, int row, int col) {
    for(int i=0;i<row;i++)
    {
        std::cout << std::endl;
        std::cout << " [ ";
        for (int j=0; j<col; j++) {
         std::cout << matrix[i * col + j] << " ";
        }
        std::cout << " ] ";
    }
    std::cout << std::endl;
}

void printMatrix(float (*matrix)[N], int row, int col) {
    for(int i=0;i<row;i++)
    {
        std::cout << std::endl;
        std::cout << " [ ";
        for (int j=0; j<col; j++) {
         std::cout << matrix[i][j] << " ";
        }
        std::cout << " ] ";
    }
    std::cout << std::endl;
}
 
int main(void)
{
        float alpha=1.0;
        float beta=1.0;
        // float h_A[M][K]={ {1,2,3}, {4,5,6} };
        // float h_B[K][N]={ {1,2,3,4}, {5,6,7,8}, {9,10,11,12} };
        // float h_C[M][N] = {0};

        float h_A[M][K]= {{1.219662, 1.467541, 10.000000, 1.000000},{7.550661, 3.893548, 17.858698, 1.000000}};
        float h_B[K][N]= {{2, 2}, {3, 1}, {1, 3}, {-12, -12}};
        float h_C[M][N] = {0};
        float result[2];
        float *d_a,*d_b,*d_c;
        float host_A[4], host_B[8];
        hipMalloc((void**)&d_a,M*K*sizeof(float));
        hipMalloc((void**)&d_b,K*N*sizeof(float));
        hipMalloc((void**)&d_c,M*N*sizeof(float));
        hipMemcpy(d_a,&h_A,M*K*sizeof(float),hipMemcpyHostToDevice);
        hipMemcpy(d_b,&h_B,K*N*sizeof(float),hipMemcpyHostToDevice);
        hipMemcpy(d_c,h_C,M*N*sizeof(float), hipMemcpyHostToDevice);

        hipMemcpy(result, d_c, M * N * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(host_A, d_a, M * K * sizeof(float), hipMemcpyDeviceToHost);
        printMatrix2(host_A, M, K);
        hipMemcpy(host_B, d_b, K * N * sizeof(float), hipMemcpyDeviceToHost);
        printMatrix2(host_B, K, N);

        hipblasHandle_t handle;
        hipblasCreate(&handle);
        printf("CHECK THE PARAM OF cublasSgemm: %d %d %d %d %d %d\n", M, N, K, K, N , M);
        hipblasSgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_N, N, M, K,&alpha, d_b, N, d_a, K,&beta, d_c, N);
        hipMemcpy(result,d_c,M*N*sizeof(float),hipMemcpyDeviceToHost);//此处的h_C是按列存储的C
        printMatrix2(result, M, N);//按行优先N行M列的顺序读取h_C相当于做了CT的结果
        return 0;
}