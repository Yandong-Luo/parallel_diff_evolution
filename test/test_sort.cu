#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define CUDA_PARAM_MAX_SIZE 16
#define T 64  // template parameter for SortParamBasedBitonic

// Error checking macro
#define CHECK_CUDA(call) { \
    hipError_t err = call; \
    if (err != hipSuccess) { \
        printf("CUDA Error: %s at line %d\n", hipGetErrorString(err), __LINE__); \
        exit(1); \
    } \
}

// Helper function to print arrays
void printArrays(float* fitness, float* params, int size) {
    printf("\nFitness values:\n");
    for(int i = 0; i < size; i++) {
        printf("%.2f ", fitness[i]);
    }
    printf("\n\nParam values (first parameter only):\n");
    for(int i = 0; i < size; i++) {
        printf("%.2f ", params[i * CUDA_PARAM_MAX_SIZE]);
    }
    printf("\n\n");
}

__device__ void BitonicWarpCompare(float &param, float &fitness, int lane_mask){
    float mapping_param = __shfl_xor_sync(0xffffffff, param, lane_mask);
    float mapping_fitness = __shfl_xor_sync(0xffffffff, fitness, lane_mask);
    // determine current sort order is increase (1.0) or decrease (-1.0)
    float sortOrder = (threadIdx.x > (threadIdx.x ^ lane_mask)) ? -1.0 : 1.0;

    if(sortOrder * (mapping_fitness - fitness) < 0.f){
        param = mapping_param;
        fitness = mapping_fitness;
    }
}

// template <int T=64>
__global__ void SortParamBasedBitonic(float *all_param, float *all_fitness){
    if (all_param == nullptr || all_fitness == nullptr) return;
    // each block have a share memory
    __shared__ float sm_sorted_fitness[T];
    __shared__ float sm_sorted_param[T];
    int param_id = blockIdx.x;
    int sol_id = threadIdx.x;
    float current_param = all_param[sol_id * CUDA_PARAM_MAX_SIZE + param_id];
    float current_fitness = all_fitness[sol_id];

    // Sort the contents of 32 threads in a warp based on Bitonic merge sort. Implement detail is the alternative representation of https://en.wikipedia.org/wiki/Bitonic_sorter
    BitonicWarpCompare(current_param, current_fitness, 1);

    BitonicWarpCompare(current_param, current_fitness, 3);
    BitonicWarpCompare(current_param, current_fitness, 1);

    BitonicWarpCompare(current_param, current_fitness, 7);
    BitonicWarpCompare(current_param, current_fitness, 2);
    BitonicWarpCompare(current_param, current_fitness, 1);

    BitonicWarpCompare(current_param, current_fitness, 15);
    BitonicWarpCompare(current_param, current_fitness, 4);
    BitonicWarpCompare(current_param, current_fitness, 2);
    BitonicWarpCompare(current_param, current_fitness, 1);

    // above all finish the sorting 16 threads in Warp, continue to finish 2 group of 16 threads
    BitonicWarpCompare(current_param, current_fitness, 31);
    BitonicWarpCompare(current_param, current_fitness, 8);
    BitonicWarpCompare(current_param, current_fitness, 4);
    BitonicWarpCompare(current_param, current_fitness, 2);
    BitonicWarpCompare(current_param, current_fitness, 1);

    // above all finsh the sort for each warp, continue to finish the sort between different warp by share memory.
    // record the warp sorting result to share memory
    sm_sorted_param[sol_id] = current_param;
    sm_sorted_fitness[sol_id] = current_fitness;

    // Wait for all thread finish above computation
    __syncthreads();

    // if T == 64 (we have 2 warp), we just need to compare these 2 warp by share memory.
    // Otherwise, we need to modify the following code

    int compare_idx = sol_id ^ 63;
    float mapping_param = sm_sorted_param[compare_idx];
    float mapping_fitness = sm_sorted_fitness[compare_idx];

    float sortOrder = (threadIdx.x > (threadIdx.x ^ 63)) ? -1.0 : 1.0;

    if(sortOrder * (mapping_fitness - current_fitness) < 0.f){
        current_param = mapping_param;
        current_fitness = mapping_fitness;
    }
    // Wait for the sort between two warp finish
    __syncthreads();
    // Now, we can come back to the sorting in the warp
    BitonicWarpCompare(current_param, current_fitness, 16);
    BitonicWarpCompare(current_param, current_fitness, 8);
    BitonicWarpCompare(current_param, current_fitness, 4);
    BitonicWarpCompare(current_param, current_fitness, 2);
    BitonicWarpCompare(current_param, current_fitness, 1);

    // above all finish all sorting for fitness and param
    if (blockIdx.x < CUDA_PARAM_MAX_SIZE){
        all_param[sol_id * CUDA_PARAM_MAX_SIZE + param_id] = current_param;
        // printf("======================== Update sorted param for solution id:%d\n", threadIdx.x);
    }
    if (blockIdx.x == 0)    all_fitness[threadIdx.x] = current_fitness;
}

int main() {
    // Host arrays
    float *h_fitness, *h_params;
    // Device arrays
    float *d_fitness, *d_params;

    CHECK_CUDA(hipSetDevice(0));
    
    // Allocate host memory
    // h_fitness = (float*)malloc(T * sizeof(float));
    // h_params = (float*)malloc(T * CUDA_PARAM_MAX_SIZE * sizeof(float));
    // 主机内存应该使用 cudaHostAlloc 而不是 malloc
    // float *h_fitness, *h_params;
    hipHostAlloc(&h_fitness, T * sizeof(float), hipHostMallocDefault);
    hipHostAlloc(&h_params, T * CUDA_PARAM_MAX_SIZE * sizeof(float), hipHostMallocDefault);
    // Initialize random seed
    srand(time(NULL));
    
    // Initialize fitness with decreasing values
    for(int i = 0; i < T; i++) {
        h_fitness[i] = (float)(T - i);  // Creates values from T down to 1
    }
    
    // Initialize params with random values
    for(int i = 0; i < T * CUDA_PARAM_MAX_SIZE; i++) {
        h_params[i] = (float)rand() / RAND_MAX * 100.0f;  // Random values between 0 and 100
    }
    
    printf("Initial arrays:");
    printArrays(h_fitness, h_params, T);
    
    // Allocate device memory
    CHECK_CUDA(hipMalloc(&d_fitness, T * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_params, T * CUDA_PARAM_MAX_SIZE * sizeof(float)));
    
    // Copy data to device
    CHECK_CUDA(hipMemcpy(d_fitness, h_fitness, T * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_params, h_params, T * CUDA_PARAM_MAX_SIZE * sizeof(float), hipMemcpyHostToDevice));

    // Launch kernel
    // We need CUDA_PARAM_MAX_SIZE blocks because we're sorting each parameter independently
    SortParamBasedBitonic<<<CUDA_PARAM_MAX_SIZE, T>>>(d_params, d_fitness);
    
    // Check for kernel launch errors
    CHECK_CUDA(hipGetLastError());
    
    // Wait for GPU to finish
    CHECK_CUDA(hipDeviceSynchronize());
    
    // Copy results back to host
    CHECK_CUDA(hipMemcpy(h_fitness, d_fitness, T * sizeof(float), hipMemcpyDeviceToHost));
    CHECK_CUDA(hipMemcpy(h_params, d_params, T * CUDA_PARAM_MAX_SIZE * sizeof(float), hipMemcpyDeviceToHost));
    
    printf("Sorted arrays:");
    printArrays(h_fitness, h_params, T);
    
    // Verify sorting
    bool sorted = true;
    for(int i = 1; i < T; i++) {
        if(h_fitness[i-1] < h_fitness[i]) {
            sorted = false;
            printf("Error: Array not properly sorted at index %d\n", i);
            break;
        }
    }
    if(sorted) {
        printf("Verification: Arrays successfully sorted in descending order!\n");
    }
    
    // Cleanup
    CHECK_CUDA(hipHostFree(h_fitness));
    CHECK_CUDA(hipHostFree(h_params));
    CHECK_CUDA(hipFree(d_fitness));
    CHECK_CUDA(hipFree(d_params));
    
    return 0;
}