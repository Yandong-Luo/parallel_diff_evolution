#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define CUDA_PARAM_MAX_SIZE 16
#define T 128  // template parameter for SortParamBasedBitonic

// Error checking macro
#define CHECK_CUDA(call) { \
    hipError_t err = call; \
    if (err != hipSuccess) { \
        printf("CUDA Error: %s at line %d\n", hipGetErrorString(err), __LINE__); \
        exit(1); \
    } \
}

// Helper function to print arrays
void printArrays(float* fitness, float* params, int size) {
    printf("\nFitness values:\n");
    for(int i = 0; i < size; i++) {
        printf("%.2f ", fitness[i]);
    }
    // printf("\n\nParam values (first parameter only):\n");
    // for(int i = 0; i < size; i++) {
    //     printf("%.2f ", params[i * CUDA_PARAM_MAX_SIZE]);
    // }
    printf("\n\n");
}

__device__ void BitonicWarpCompare(float &param, float &fitness, int lane_mask){
    float mapping_param = __shfl_xor_sync(0xffffffff, param, lane_mask);
    float mapping_fitness = __shfl_xor_sync(0xffffffff, fitness, lane_mask);
    // determine current sort order is increase (1.0) or decrease (-1.0)
    float sortOrder = (threadIdx.x > (threadIdx.x ^ lane_mask)) ? -1.0 : 1.0;

    if(sortOrder * (mapping_fitness - fitness) < 0.f){
        param = mapping_param;
        fitness = mapping_fitness;
    }
}

// template <int T=64>
// sort 64
__global__ void SortParamBasedBitonic(float *all_param, float *all_fitness){
    if (all_param == nullptr || all_fitness == nullptr) return;
    // each block have a share memory
    __shared__ float sm_sorted_fitness[T];
    __shared__ float sm_sorted_param[T];
    int param_id = blockIdx.x;
    int sol_id = threadIdx.x;
    float current_param = all_param[sol_id * CUDA_PARAM_MAX_SIZE + param_id];
    float current_fitness = all_fitness[sol_id];
    int compare_idx;
    float mapping_param, mapping_fitness, sortOrder;

    if (threadIdx.x <64){
        // Sort the contents of 32 threads in a warp based on Bitonic merge sort. Implement detail is the alternative representation of https://en.wikipedia.org/wiki/Bitonic_sorter
        BitonicWarpCompare(current_param, current_fitness, 1);

        BitonicWarpCompare(current_param, current_fitness, 3);
        BitonicWarpCompare(current_param, current_fitness, 1);

        BitonicWarpCompare(current_param, current_fitness, 7);
        BitonicWarpCompare(current_param, current_fitness, 2);
        BitonicWarpCompare(current_param, current_fitness, 1);

        BitonicWarpCompare(current_param, current_fitness, 15);
        BitonicWarpCompare(current_param, current_fitness, 4);
        BitonicWarpCompare(current_param, current_fitness, 2);
        BitonicWarpCompare(current_param, current_fitness, 1);

        // above all finish the sorting 16 threads in Warp, continue to finish 2 group of 16 threads
        BitonicWarpCompare(current_param, current_fitness, 31);
        BitonicWarpCompare(current_param, current_fitness, 8);
        BitonicWarpCompare(current_param, current_fitness, 4);
        BitonicWarpCompare(current_param, current_fitness, 2);
        BitonicWarpCompare(current_param, current_fitness, 1);

        // above all finsh the sort for each warp, continue to finish the sort between different warp by share memory.
        // record the warp sorting result to share memory
        sm_sorted_param[sol_id] = current_param;
        sm_sorted_fitness[sol_id] = current_fitness;
    }
    
    // Wait for all thread finish above computation
    __syncthreads();

    if(threadIdx.x < 64){
        // if T == 64 (we have 2 warp), we just need to compare these 2 warp by share memory.
        // Otherwise, we need to modify the following code

        compare_idx = sol_id ^ 63;
        mapping_param = sm_sorted_param[compare_idx];
        mapping_fitness = sm_sorted_fitness[compare_idx];

        sortOrder = (threadIdx.x > (threadIdx.x ^ 63)) ? -1.0 : 1.0;

        if(sortOrder * (mapping_fitness - current_fitness) < 0.f){
            current_param = mapping_param;
            current_fitness = mapping_fitness;
        }
    }

    
    // Wait for the sort between two warp finish
    __syncthreads();

    if(threadIdx.x < 64){
        // Now, we can come back to the sorting in the warp
        BitonicWarpCompare(current_param, current_fitness, 16);
        BitonicWarpCompare(current_param, current_fitness, 8);
        BitonicWarpCompare(current_param, current_fitness, 4);
        BitonicWarpCompare(current_param, current_fitness, 2);
        BitonicWarpCompare(current_param, current_fitness, 1);

        // above all finish all sorting for fitness and param
        if (blockIdx.x < CUDA_PARAM_MAX_SIZE){
            all_param[sol_id * CUDA_PARAM_MAX_SIZE + param_id] = current_param;
            // printf("======================== Update sorted param for solution id:%d\n", threadIdx.x);
        }
        if (blockIdx.x == 0)    all_fitness[threadIdx.x] = current_fitness;
    }
    
}

// Sort only the last half of the elements in ascending order (128)
__global__ void SortParamBasedBitonic2(float *all_param, float *all_fitness){
    if (all_param == nullptr || all_fitness == nullptr) return;
    // each block have a share memory
    __shared__ float sm_sorted_fitness[T];
    __shared__ float sm_sorted_param[T];
    int param_id = blockIdx.x;
    int sol_id = threadIdx.x;
    float current_param = all_param[sol_id * CUDA_PARAM_MAX_SIZE + param_id + 64];
    float current_fitness = all_fitness[sol_id+64];

    int compare_idx;
    float mapping_param, mapping_fitness, sortOrder;

    if (threadIdx.x >= 64){
        // Sort the contents of 32 threads in a warp based on Bitonic merge sort. Implement detail is the alternative representation of https://en.wikipedia.org/wiki/Bitonic_sorter
        BitonicWarpCompare(current_param, current_fitness, 1);

        BitonicWarpCompare(current_param, current_fitness, 3);
        BitonicWarpCompare(current_param, current_fitness, 1);

        BitonicWarpCompare(current_param, current_fitness, 7);
        BitonicWarpCompare(current_param, current_fitness, 2);
        BitonicWarpCompare(current_param, current_fitness, 1);

        BitonicWarpCompare(current_param, current_fitness, 15);
        BitonicWarpCompare(current_param, current_fitness, 4);
        BitonicWarpCompare(current_param, current_fitness, 2);
        BitonicWarpCompare(current_param, current_fitness, 1);

        // above all finish the sorting 16 threads in Warp, continue to finish 2 group of 16 threads
        BitonicWarpCompare(current_param, current_fitness, 31);
        BitonicWarpCompare(current_param, current_fitness, 8);
        BitonicWarpCompare(current_param, current_fitness, 4);
        BitonicWarpCompare(current_param, current_fitness, 2);
        BitonicWarpCompare(current_param, current_fitness, 1);

        // above all finsh the sort for each warp, continue to finish the sort between different warp by share memory.
        // record the warp sorting result to share memory
        sm_sorted_param[sol_id ] = current_param;
        sm_sorted_fitness[sol_id] = current_fitness;
    }
    
    // Wait for all thread finish above computation
    __syncthreads();

    if (threadIdx.x >= 64)
    {
        compare_idx = sol_id ^ 63;
        mapping_param = sm_sorted_param[compare_idx];
        mapping_fitness = sm_sorted_fitness[compare_idx];

        sortOrder = (threadIdx.x > (threadIdx.x ^ 63)) ? -1.0 : 1.0;

        if(sortOrder * (mapping_fitness - current_fitness) < 0.f){
            current_param = mapping_param;
            current_fitness = mapping_fitness;
        }
    }
    
    
    // Wait for the sort between two warp finish
    __syncthreads();
    if(threadIdx.x >= 64){
        // Now, we can come back to the sorting in the warp
        BitonicWarpCompare(current_param, current_fitness, 16);
        BitonicWarpCompare(current_param, current_fitness, 8);
        BitonicWarpCompare(current_param, current_fitness, 4);
        BitonicWarpCompare(current_param, current_fitness, 2);
        BitonicWarpCompare(current_param, current_fitness, 1);

        sm_sorted_param[sol_id ] = current_param;
        sm_sorted_fitness[sol_id] = current_fitness;
    }
    
    __syncthreads();
    if(threadIdx.x >= 64){
        compare_idx = threadIdx.x ^ 127;
        mapping_param = sm_sorted_param[compare_idx];
        mapping_fitness = sm_sorted_fitness[compare_idx];
        sortOrder = (threadIdx.x > compare_idx) ? -1.f : 1.f;

        if (sortOrder * (mapping_fitness - current_fitness) < 0.f) {
            current_fitness = mapping_fitness;
            current_param = mapping_param;
            sm_sorted_fitness[threadIdx.x] = current_fitness;
            sm_sorted_param[threadIdx.x] = current_param;
        }
    }
    
    __syncthreads();
    if(threadIdx.x >= 64){
        compare_idx = threadIdx.x ^ 32;
        mapping_fitness = sm_sorted_fitness[compare_idx];
        mapping_param = sm_sorted_param[compare_idx];
        sortOrder = (threadIdx.x > compare_idx) ? -1.f : 1.f;
        if (sortOrder * (mapping_fitness - current_fitness) < 0.f) {
            current_fitness = mapping_fitness;
            current_param = mapping_param;
        }
        BitonicWarpCompare(current_param, current_fitness, 16);
        BitonicWarpCompare(current_param, current_fitness, 8);
        BitonicWarpCompare(current_param, current_fitness, 4);
        BitonicWarpCompare(current_param, current_fitness, 2);
        BitonicWarpCompare(current_param, current_fitness, 1);

        // above all finish all sorting for fitness and param
        if (blockIdx.x < CUDA_PARAM_MAX_SIZE){
            all_param[sol_id * CUDA_PARAM_MAX_SIZE + param_id+64] = current_param;
            // printf("======================== Update sorted param for solution id:%d\n", threadIdx.x);
        }
        if (blockIdx.x == 0)    all_fitness[threadIdx.x+64] = current_fitness;
    }
}

// Sort only the last half of the elements in ascending order (128)
__global__ void SortParamBasedBitonic3(float *all_param, float *all_fitness){
    if (all_param == nullptr || all_fitness == nullptr) return;
    // each block have a share memory
    __shared__ float sm_sorted_fitness[T];
    __shared__ float sm_sorted_param[T];
    int param_id = blockIdx.x;
    int sol_id = threadIdx.x;
    float current_param;
    float current_fitness;

    if(threadIdx.x < 64){
        current_param = all_param[sol_id * CUDA_PARAM_MAX_SIZE + param_id];
        current_fitness = all_fitness[sol_id];
    }
    else{
        current_param = all_param[sol_id * CUDA_PARAM_MAX_SIZE + param_id + 64];
        current_fitness = all_fitness[sol_id+64];
    }
     

    int compare_idx;
    float mapping_param, mapping_fitness, sortOrder;

    // if (threadIdx.x >= 64){
        // Sort the contents of 32 threads in a warp based on Bitonic merge sort. Implement detail is the alternative representation of https://en.wikipedia.org/wiki/Bitonic_sorter
        BitonicWarpCompare(current_param, current_fitness, 1);

        BitonicWarpCompare(current_param, current_fitness, 3);
        BitonicWarpCompare(current_param, current_fitness, 1);

        BitonicWarpCompare(current_param, current_fitness, 7);
        BitonicWarpCompare(current_param, current_fitness, 2);
        BitonicWarpCompare(current_param, current_fitness, 1);

        BitonicWarpCompare(current_param, current_fitness, 15);
        BitonicWarpCompare(current_param, current_fitness, 4);
        BitonicWarpCompare(current_param, current_fitness, 2);
        BitonicWarpCompare(current_param, current_fitness, 1);

        // above all finish the sorting 16 threads in Warp, continue to finish 2 group of 16 threads
        BitonicWarpCompare(current_param, current_fitness, 31);
        BitonicWarpCompare(current_param, current_fitness, 8);
        BitonicWarpCompare(current_param, current_fitness, 4);
        BitonicWarpCompare(current_param, current_fitness, 2);
        BitonicWarpCompare(current_param, current_fitness, 1);

        // above all finsh the sort for each warp, continue to finish the sort between different warp by share memory.
        // record the warp sorting result to share memory
        sm_sorted_param[sol_id ] = current_param;
        sm_sorted_fitness[sol_id] = current_fitness;
    // }
    
    // Wait for all thread finish above computation
    __syncthreads();

    // if (threadIdx.x >= 64)
    // {
        compare_idx = sol_id ^ 63;
        mapping_param = sm_sorted_param[compare_idx];
        mapping_fitness = sm_sorted_fitness[compare_idx];

        sortOrder = (threadIdx.x > (threadIdx.x ^ 63)) ? -1.0 : 1.0;

        if(sortOrder * (mapping_fitness - current_fitness) < 0.f){
            current_param = mapping_param;
            current_fitness = mapping_fitness;
        }
    // }
    
    
    // Wait for the sort between two warp finish
    __syncthreads();
    // if(threadIdx.x >= 64){
        // Now, we can come back to the sorting in the warp
        BitonicWarpCompare(current_param, current_fitness, 16);
        BitonicWarpCompare(current_param, current_fitness, 8);
        BitonicWarpCompare(current_param, current_fitness, 4);
        BitonicWarpCompare(current_param, current_fitness, 2);
        BitonicWarpCompare(current_param, current_fitness, 1);

        sm_sorted_param[sol_id ] = current_param;
        sm_sorted_fitness[sol_id] = current_fitness;
    // }
    
    __syncthreads();
    if(threadIdx.x >= 64){
        compare_idx = threadIdx.x ^ 127;
        mapping_param = sm_sorted_param[compare_idx];
        mapping_fitness = sm_sorted_fitness[compare_idx];
        sortOrder = (threadIdx.x > compare_idx) ? -1.f : 1.f;

        if (sortOrder * (mapping_fitness - current_fitness) < 0.f) {
            current_fitness = mapping_fitness;
            current_param = mapping_param;
            sm_sorted_fitness[threadIdx.x] = current_fitness;
            sm_sorted_param[threadIdx.x] = current_param;
        }
    }
    
    __syncthreads();
    if(threadIdx.x >= 64){
        compare_idx = threadIdx.x ^ 32;
        mapping_fitness = sm_sorted_fitness[compare_idx];
        mapping_param = sm_sorted_param[compare_idx];
        sortOrder = (threadIdx.x > compare_idx) ? -1.f : 1.f;
        if (sortOrder * (mapping_fitness - current_fitness) < 0.f) {
            current_fitness = mapping_fitness;
            current_param = mapping_param;
        }
        BitonicWarpCompare(current_param, current_fitness, 16);
        BitonicWarpCompare(current_param, current_fitness, 8);
        BitonicWarpCompare(current_param, current_fitness, 4);
        BitonicWarpCompare(current_param, current_fitness, 2);
        BitonicWarpCompare(current_param, current_fitness, 1);
    }

    if(threadIdx.x >= 64){
        // above all finish all sorting for fitness and param
        if (blockIdx.x < CUDA_PARAM_MAX_SIZE){
            all_param[sol_id * CUDA_PARAM_MAX_SIZE + param_id+64] = current_param;
            // printf("======================== Update sorted param for solution id:%d\n", threadIdx.x);
        }
        if (blockIdx.x == 0)    all_fitness[threadIdx.x+64] = current_fitness;
    }
    else{
        // above all finish all sorting for fitness and param
        if (blockIdx.x < CUDA_PARAM_MAX_SIZE){
            all_param[sol_id * CUDA_PARAM_MAX_SIZE + param_id] = current_param;
            // printf("======================== Update sorted param for solution id:%d\n", threadIdx.x);
        }
        if (blockIdx.x == 0)    all_fitness[threadIdx.x] = current_fitness;
    }
}

int main() {
    // Host arrays
    float *h_fitness, *h_params;
    // Device arrays
    float *d_fitness, *d_params;

    CHECK_CUDA(hipSetDevice(0));
    
    // Allocate host memory
    hipHostAlloc(&h_fitness, T * sizeof(float), hipHostMallocDefault);
    hipHostAlloc(&h_params, T * CUDA_PARAM_MAX_SIZE * sizeof(float), hipHostMallocDefault);
    // Initialize random seed
    srand(time(NULL));
    printf("T:%d\n", T);
    // Initialize fitness with decreasing values
    for(int i = 0; i < T; i++) {
        h_fitness[i] = (float)(T - i);  // Creates values from T down to 1
    }
    
    // Initialize params with random values
    for(int i = 0; i < T * CUDA_PARAM_MAX_SIZE; i++) {
        h_params[i] = (float)rand() / RAND_MAX * 100.0f;  // Random values between 0 and 100
    }
    
    printf("Initial arrays:");
    printArrays(h_fitness, h_params, T);
    
    // Allocate device memory
    CHECK_CUDA(hipMalloc(&d_fitness, T * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_params, T * CUDA_PARAM_MAX_SIZE * sizeof(float)));
    
    // Copy data to device
    CHECK_CUDA(hipMemcpy(d_fitness, h_fitness, T * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_params, h_params, T * CUDA_PARAM_MAX_SIZE * sizeof(float), hipMemcpyHostToDevice));

    // Launch kernel
    // We need CUDA_PARAM_MAX_SIZE blocks because we're sorting each parameter independently
    // SortParamBasedBitonic<<<CUDA_PARAM_MAX_SIZE, T>>>(d_params, d_fitness);
    
    // SortParamBasedBitonic2<<<CUDA_PARAM_MAX_SIZE, T>>>(d_params, d_fitness);

    SortParamBasedBitonic3<<<CUDA_PARAM_MAX_SIZE, T>>>(d_params, d_fitness);

    
    // Check for kernel launch errors
    CHECK_CUDA(hipGetLastError());
    
    // Wait for GPU to finish
    CHECK_CUDA(hipDeviceSynchronize());
    
    // Copy results back to host
    CHECK_CUDA(hipMemcpy(h_fitness, d_fitness, T * sizeof(float), hipMemcpyDeviceToHost));
    CHECK_CUDA(hipMemcpy(h_params, d_params, T * CUDA_PARAM_MAX_SIZE * sizeof(float), hipMemcpyDeviceToHost));
    
    printf("Sorted arrays:");
    printArrays(h_fitness, h_params, T);
    
    // Verify sorting
    bool sorted = true;
    for(int i = 1; i < T; i++) {
        if(h_fitness[i-1] < h_fitness[i]) {
            sorted = false;
            printf("Error: Array not properly sorted at index %d\n", i);
            break;
        }
    }
    if(sorted) {
        printf("Verification: Arrays successfully sorted in descending order!\n");
    }
    
    // Cleanup
    CHECK_CUDA(hipHostFree(h_fitness));
    CHECK_CUDA(hipHostFree(h_params));
    CHECK_CUDA(hipFree(d_fitness));
    CHECK_CUDA(hipFree(d_params));
    
    return 0;
}