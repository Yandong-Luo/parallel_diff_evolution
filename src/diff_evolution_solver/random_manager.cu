#include "diff_evolution_solver/random_manager.cuh"

namespace cudaprocess{
    CudaRandomManager::CudaRandomManager(int gpu_device) {
        CHECK_CUDA(hipSetDevice(gpu_device));
        CHECK_CUDA(hipStreamCreate(&stream));
        
        CHECK_CUDA(hipMalloc(&uniform_data_, sizeof(float) * uniform_size_));
        CHECK_CUDA(hipMalloc(&normal_data_, sizeof(float) * normal_size_));
        
        CURAND_CHECK(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_XORWOW));
        CURAND_CHECK(hiprandSetStream(gen, stream));
        CURAND_CHECK(hiprandSetPseudoRandomGeneratorSeed(gen, 0));
        
        Generate();
    }

    CudaRandomManager::~CudaRandomManager() {
        hipStreamSynchronize(stream);
        CURAND_CHECK(hiprandDestroyGenerator(gen));
        CHECK_CUDA(hipFree(uniform_data_));
        CHECK_CUDA(hipFree(normal_data_));
        CHECK_CUDA(hipStreamDestroy(stream));
    }

    void CudaRandomManager::Generate() {
        CURAND_CHECK(hiprandGenerateUniform(gen, uniform_data_, uniform_size_));
        CURAND_CHECK(hiprandGenerateNormal(gen, normal_data_, normal_size_, 0.0f, 1.0f));
        hipStreamSynchronize(stream);
    }
}