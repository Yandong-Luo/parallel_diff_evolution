#include "hip/hip_runtime.h"
#include "diff_evolution_solver/solver.cuh"
#include "diff_evolution_solver/decoder.cuh"
#include "diff_evolution_solver/debug.cuh"
#include "diff_evolution_solver/evolve.cuh"
#include "diff_evolution_solver/evaluate.cuh"
#include "utils/utils_fun.cuh"
#include <math.h>

namespace cudaprocess{

void CudaDiffEvolveSolver::MallocSetup(){
    CHECK_CUDA(hipSetDevice(gpu_device_));

    // GPU Device
    // CHECK_CUDA(hipMalloc(&decoder_, sizeof(CudaProblemDecoder)));
    CHECK_CUDA(hipMalloc(&evolve_data_, sizeof(CudaEvolveData)));
    CHECK_CUDA(hipMalloc(&new_cluster_data_, sizeof(CudaParamClusterData<64>)));
    CHECK_CUDA(hipMalloc(&old_cluster_data_, sizeof(CudaParamClusterData<192>)));
    // CHECK_CUDA(hipMalloc(&new_cluster_vec_, sizeof(CudaVector<CudaParamIndividual, CUDA_SOLVER_POP_SIZE>)));
    // CHECK_CUDA(hipMalloc(&problem_, sizeof(Problem)));
    CHECK_CUDA(hipMalloc(&evaluate_score_, CUDA_SOLVER_POP_SIZE * sizeof(float)));
    CHECK_CUDA(hipMalloc(&last_fitness, sizeof(float)));
    CHECK_CUDA(hipMalloc(&terminate_flag, sizeof(int)));
    CHECK_CUDA(hipMalloc(&result, sizeof(CudaParamIndividual)));

    // objective, constraint, tmp_score, lambda, parameter matrix
    CHECK_CUDA(hipMalloc(&constraint_matrix, row_constraint * col_constraint * sizeof(float)));
    CHECK_CUDA(hipMalloc(&objective_matrix, row_obj * col_obj * sizeof(float)));
    CHECK_CUDA(hipMalloc(&tmp_score, CUDA_SOLVER_POP_SIZE * row_constraint * sizeof(float)));
    CHECK_CUDA(hipMalloc(&lambda_matrix, row_lambda * col_lambda * sizeof(float)));
    CHECK_CUDA(hipMalloc(&param_matrix, (dims_ + 1) * CUDA_SOLVER_POP_SIZE * sizeof(float)));

    // CPU Host
    CHECK_CUDA(hipHostAlloc(&h_terminate_flag, sizeof(int), hipHostMallocDefault));
    CHECK_CUDA(hipHostAlloc(&host_result, sizeof(CudaParamIndividual), hipHostMallocDefault));
    CHECK_CUDA(hipHostAlloc(&host_evolve_data_, sizeof(CudaEvolveData), hipHostMallocDefault));

    if (DEBUG_PRINT_FLAG || DEBUG_PRINT_SOLVER_FLAG){
        CHECK_CUDA(hipHostAlloc(&host_new_cluster_data_, sizeof(CudaParamClusterData<64>), hipHostMallocDefault));
        CHECK_CUDA(hipHostAlloc(&host_old_cluster_data_, sizeof(CudaParamClusterData<192>), hipHostMallocDefault));
    }
    
    if (DEBUG_PRINT_FLAG || DEBUG_PRINT_EVALUATE_FLAG){
        // objective, constraint, tmp_score, lambda, parameter, score matrix
        CHECK_CUDA(hipHostAlloc(&h_constraint_matrix, row_constraint * col_constraint * sizeof(float), hipHostMallocDefault));
        CHECK_CUDA(hipHostAlloc(&h_objective_matrix, row_obj * col_obj * sizeof(float), hipHostMallocDefault));
        CHECK_CUDA(hipHostAlloc(&host_tmp_score, CUDA_SOLVER_POP_SIZE * row_constraint * sizeof(float), hipHostMallocDefault));
        CHECK_CUDA(hipHostAlloc(&h_lambda_matrix, row_lambda * col_lambda * sizeof(float), hipHostMallocDefault));
        CHECK_CUDA(hipHostAlloc(&host_param_matrix, (dims_ + 1) * CUDA_SOLVER_POP_SIZE * sizeof(float), hipHostMallocDefault));

        CHECK_CUDA(hipHostAlloc(&host_evaluate_score_, CUDA_SOLVER_POP_SIZE * sizeof(float), hipHostMallocDefault));
    }


    cuda_utils_ = std::make_shared<CudaUtil>();

    cudamalloc_flag = true;
}

__global__ void ResetEvolveData(CudaEvolveData* evolve) {
    // 重置LSHADE参数
    evolve->hist_lshade_param = {0.6f, 0.6f, 0.9f, 0.0f};  // 默认初始值
    
    // 清空last_potential_sol
    evolve->last_potential_sol.len = 0;
    
    // 重置warm_start
    evolve->warm_start.fitness = CUDA_MAX_FLOAT;
    
    // 重置problem参数
    evolve->problem_param.top_ratio = 0.0f;
}

template <int T>
__global__ void ResetClusterData(CudaParamClusterData<T>* data) {
    int idx = threadIdx.x;
    if (idx == 0) {
        data->len = 0;
    }
    if (idx < T) {
        data->fitness[idx] = CUDA_MAX_FLOAT;
    }
}

void CudaDiffEvolveSolver::InitDiffEvolveParam(float top, float d_top, float min_top, float diff, float d_diff, float min_diff, float scale_f, float prob_crossover){
    top_ = top;
    d_top_ = d_top;
    min_top_ = min_top;
    diff_ = diff;
    d_diff_ = d_diff;
    min_diff_ = min_diff;
    
    lshade_param_.scale_f = lshade_param_.scale_f1 = scale_f;
    lshade_param_.Cr = prob_crossover;

}

__global__ void InitCudaEvolveData(CudaEvolveData* evolve, CudaParamClusterData<192>* old_cluster_data, int pop_size){
    int idx = threadIdx.x;
    if (idx == 0) {
        evolve->problem_param.top_ratio = 0.;
        evolve->hist_lshade_param.scale_f = evolve->hist_lshade_param.scale_f1 = 0.6;
        evolve->hist_lshade_param.Cr = 0.9;
        // evolve->new_cluster_vec->len = pop_size;
        old_cluster_data->len = pop_size;
    }
    if (idx < pop_size){
        // initial the each parameter in old_cluster 
        for (int i = 0; i < CUDA_PARAM_MAX_SIZE; ++i){
            old_cluster_data->all_param[(idx + pop_size) * CUDA_PARAM_MAX_SIZE + i] = 0.f;
        }
        old_cluster_data->fitness[idx + pop_size] = CUDA_MAX_FLOAT;
    }
}

void CudaDiffEvolveSolver::SetBoundary(Problem* problem){
    for (int i = 0; i < con_var_dims_; ++i){
        host_evolve_data_->upper_bound[i] = host_upper_bound_[i] = problem->con_upper_bound[i];
        host_evolve_data_->lower_bound[i] = host_lower_bound_[i] = problem->con_lower_bound[i];
    }

    for (int i = con_var_dims_, j = 0; i < dims_; ++i, ++j){
        host_evolve_data_->upper_bound[i] = host_upper_bound_[i] = problem->int_upper_bound[j];
        host_evolve_data_->lower_bound[i] = host_lower_bound_[i] = problem->int_lower_bound[j];
    }
    // for(int i = 0; i < host_decoder_->dims_; ++i){
    //     printf("index:%d lower bound:%f, upper bound:%f\n",i, host_evolve_data_->lower_bound[i], host_evolve_data_->upper_bound[i]);
    // }
}

/**
 * CudaEvolveData* ----> CudaParamClusterData<T> *
 */
__global__ void UpdateClusterDataBasedEvolve(CudaEvolveData* evolve_data, CudaParamClusterData<64>* new_cluster_data, int num_last_potential_sol){
    int idx = blockIdx.x;
    if (idx >= num_last_potential_sol)   return;
    ConvertCudaParam<64>(new_cluster_data, &evolve_data->last_potential_sol.data[idx], idx, threadIdx.x);
}

/**
 * CudaParamClusterData ----> CudaParamIndividual * as output
 */
template <int T>
__global__ void UpdateVecParamBasedClusterData(CudaParamIndividual *output, CudaParamClusterData<T> *cluster_data){
    ConvertCudaParamRevert<T>(cluster_data, &output[blockIdx.x], blockIdx.x, threadIdx.x);
}

/**
 * CudaParamClusterData<T> * ---->  CudaEvolveData* 
 */
__global__ void UpdateEvolveWarmStartBasedClusterData(CudaEvolveData *evolve_data, CudaParamClusterData<192> *old_param){
    ConvertCudaParamRevert<192>(old_param, &evolve_data->warm_start, 0, threadIdx.x);
}

__global__ void SaveNewParamAsOldParam(CudaParamClusterData<64> *new_param, CudaParamClusterData<192> *old_param, int left_bound, int right_bound, int bias){
    int sol_id = blockIdx.x;
    if (sol_id < left_bound || sol_id >= right_bound)   return;
    ConvertCudaParamBetweenClusters<64, 192>(new_param, old_param, sol_id, sol_id + bias, threadIdx.x);
}

__global__ void GenerativeRandSolNearBest(CudaEvolveData *evolve, CudaParamClusterData<64> *new_param, float *uniform_data, int rand_idx, float delta_con, float delta_int, int size){
    int sol_id = blockIdx.x;
    int param_id = threadIdx.x;

    if (sol_id == 0 || sol_id >= size)  return;
    float upper_bound = evolve->upper_bound[param_id];
    float lower_bound = evolve->lower_bound[param_id];

    if (param_id < evolve->problem_param.con_var_dims){
        float rand_range = (upper_bound - lower_bound) * delta_con;

        // based on rand_range update the boundary
        upper_bound = min(upper_bound, new_param->all_param[param_id] + rand_range);
        lower_bound = max(lower_bound, new_param->all_param[param_id] - rand_range);
        
        // based on new boundary near parameter to generate the new parameter
        new_param->all_param[sol_id * CUDA_PARAM_MAX_SIZE + param_id] = lower_bound + uniform_data[64 * 100 * CUDA_PARAM_MAX_SIZE + sol_id * CUDA_SOLVER_POP_SIZE + rand_idx + param_id] * (upper_bound - lower_bound);
    }
    else if(param_id < evolve->problem_param.int_var_dims){
        float rand_range = (upper_bound - lower_bound) * delta_int;

        // based on rand_range update the boundary
        upper_bound = min(upper_bound, new_param->all_param[param_id] + rand_range);
        lower_bound = max(lower_bound, new_param->all_param[param_id] - rand_range);
        
        // based on new boundary near parameter to generate the new parameter
        new_param->all_param[sol_id * CUDA_PARAM_MAX_SIZE + param_id] = lower_bound + uniform_data[64 * 100 * CUDA_PARAM_MAX_SIZE + sol_id * CUDA_SOLVER_POP_SIZE + rand_idx + param_id] * (upper_bound - lower_bound);
    }
}


void CudaDiffEvolveSolver::WarmStart(Problem* problem, CudaParamIndividual* output_sol){
    InitParameter<<<1, CUDA_SOLVER_POP_SIZE, 0, cuda_utils_->streams_[task_id_]>>>(evolve_data_, CUDA_SOLVER_POP_SIZE, new_cluster_data_, old_cluster_data_, random_center_->uniform_data_);
    // CHECK_CUDA(hipStreamSynchronize(cuda_utils_->streams_[task_id_]));
    // if(last_potential_sol_.len > 0){
    //     if (DEBUG_PRINT_FLAG || DEBUG_PRINT_INIT_SOLVER_FLAG) printf("USING LAST POTENTIAL SOL\n");
    //     // int half_pop_size = CUDA_SOLVER_POP_SIZE >> 1;
    //     int quad_pop_size = CUDA_SOLVER_POP_SIZE >> 2;
    //     // one cluster generate one solution, each cluster works on one block. 
    //     // We need to generate quad_pop_size new solutions based on last potential solution, so init the new cluster in quad_pop_size grid.
    //     UpdateClusterDataBasedEvolve<<<quad_pop_size, CUDA_PARAM_MAX_SIZE, 0, cuda_utils_->streams_[task_id_]>>>(evolve_data_, new_cluster_data_, last_potential_sol_.len);
    // }
    // UpdateVecParamBasedClusterData<64><<<CUDA_SOLVER_POP_SIZE, 16, 0, cuda_utils_->streams_[task_id_]>>>(new_cluster_vec_->data, new_cluster_data_);

    // int cet = 10;
    // Update the output param based on warm start.
    // CHECK_CUDA(hipMemcpyAsync(output_sol, &new_cluster_vec_->data[cet], sizeof(CudaParamIndividual), hipMemcpyDeviceToHost, cuda_utils_->streams_[task_id_]));

    // Evaluate random solutions or potential solutions in warmstart
    Evaluation(CUDA_SOLVER_POP_SIZE, 0);

    // SortParamBasedBitonic<64><<<16, 64, 0, cuda_utils_->streams_[task_id_]>>>(new_cluster_data_->all_param, new_cluster_data_->fitness);

    // Find the best solution among the random solutions or potential solutions in warmstart and put it in the first place
    ParaFindMax2<CUDA_SOLVER_POP_SIZE, 64><<<1, CUDA_SOLVER_POP_SIZE, 0, cuda_utils_->streams_[task_id_]>>>(new_cluster_data_);

    printf("CUDA_SOLVER_POP_SIZE:%d\n", CUDA_SOLVER_POP_SIZE);
    // based on warm start result to generate random solution. Further improve the quality of the initial population
    GenerativeRandSolNearBest<<<CUDA_SOLVER_POP_SIZE, CUDA_PARAM_MAX_SIZE, 0, cuda_utils_->streams_[task_id_]>>>(evolve_data_, new_cluster_data_, random_center_->uniform_data_, 16, 0.1, 0.1, CUDA_SOLVER_POP_SIZE);

    // convert the parameter from warm start to old parameter
    SaveNewParamAsOldParam<<<CUDA_SOLVER_POP_SIZE, CUDA_PARAM_MAX_SIZE, 0, cuda_utils_->streams_[task_id_]>>>(new_cluster_data_, old_cluster_data_, 0, CUDA_SOLVER_POP_SIZE, 0);

    // Based on all old parameter to update the warm start of evolve data
    // 将 old_cluster_data_<192> 中索引为0的数据提取出来,填充到evolve data单个CudaParamIndividual结构中,记为warm start。索引为0的解是warm start过程中最优的
    UpdateEvolveWarmStartBasedClusterData<<<1, CUDA_PARAM_MAX_SIZE, 0, cuda_utils_->streams_[task_id_]>>>(evolve_data_, old_cluster_data_);

    // CHECK_CUDA(hipStreamSynchronize(cuda_utils_->streams_[task_id_]));
}

// (Abandoned) Use for loop to evaluate 
// template<int T>
// __global__ void MainEvaluation(CudaEvolveData *evolve, CudaParamClusterData<T> *cluster_data){
//     DynamicEvaluation2(evolve, cluster_data, evolve->lambda);
// }

void CudaDiffEvolveSolver::Evaluation(int size, int epoch){
    // Row-major arrangement (size x dims+1 matrix)
    // ConvertClusterToMatrix<64><<<size, dims_ + 1, 0, cuda_utils_->streams_[task_id_]>>>(evolve_data_, new_cluster_data_, param_matrix);

    // row-major arrangement (dims+1 x size matrix)
    ConvertClusterToMatrix2<64><<<dims_ + 1, size, 0, cuda_utils_->streams_[task_id_]>>>(evolve_data_, new_cluster_data_, param_matrix, size);

    // printf("device obj_constant_matrix\n");
    // printMatrix<<<1, row_obj_constant*col_obj_constant, 0, cuda_utils_->streams_[task_id_]>>>(obj_constant_matrix);

    // printf("device objective_matrix\n");
    // printMatrix<<<1, row_obj*col_obj, 0, cuda_utils_->streams_[task_id_]>>>(objective_matrix);
    
    float alpha = 1.;
    float beta = 1.;

    // reset the evaluate score and tmp score
    hipMemset(evaluate_score_, 0, size * sizeof(float));

    hipMemset(tmp_score, 0, size * row_constraint * sizeof(float));

    // Based on current epoch and interpolation to update lambda
    UpdateLambdaBasedInterpolation<<<1, row_lambda * col_lambda, 0, cuda_utils_->streams_[task_id_]>>>(evolve_data_, lambda_matrix, epoch);

    // checking before matrix multiplication
    if(DEBUG_PRINT_FLAG || DEBUG_PRINT_EVALUATE_FLAG){
        
        CHECK_CUDA(hipMemcpy(host_param_matrix, param_matrix, (dims_+1) * size * sizeof(float), hipMemcpyDeviceToHost));
        CHECK_CUDA(hipMemcpy(h_objective_matrix, objective_matrix, row_obj * col_obj * sizeof(float), hipMemcpyDeviceToHost));
        CHECK_CUDA(hipMemcpy(h_constraint_matrix, constraint_matrix, row_constraint * col_constraint * sizeof(float), hipMemcpyDeviceToHost));
        CHECK_CUDA(hipMemcpy(h_lambda_matrix, lambda_matrix, row_lambda * col_lambda * sizeof(float), hipMemcpyDeviceToHost));

        CHECK_CUDA(hipStreamSynchronize(cuda_utils_->streams_[task_id_]));

        // check parameter matrix before matrix multiplication
        PrintMatrixByRow(host_param_matrix,  dims_+1, size, "CHECK PARAMETER MATRIX BEFORE MATRIX MULTIPLICATION");

        // check obj matrix before matrix multiplication
        PrintMatrixByRow(h_objective_matrix, row_obj, col_obj, "CHECK OBJ MATRIX BEFORE MATRIX MULTIPLICATION");

        // check constraint matrix before matrix multiplication
        PrintMatrixByRow(h_constraint_matrix, row_constraint, col_constraint, "CHECK CONSTRAINT MATRIX BEFORE MATRIX MULTIPLICATION");

        // check lambda matrix before matrix multiplication
        PrintMatrixByRow(h_lambda_matrix, row_lambda, col_lambda, "CHECK LAMBDA MATRIX BEFORE MATRIX MULTIPLICATION");
    }

    // example:
    // param_matrix: pop_size x dims (64 x 3)
    // objective_matrix: 3 x 1
    // obj_constant_matrix: 64 x 1
    //     hipblasSgemm(
    //     handle,
    //     HIPBLAS_OP_T,  // A is stored row-first, so it is considered transposed, otherwise HIPBLAS_OP_N
    //     HIPBLAS_OP_T,  // B is stored in row-major order, so it is considered transposed, otherwise HIPBLAS_OP_N
    //     m, n, k,     // m: the number of rows in C; n: the number of columns in C; k: the number of columns in A; 
    //     &alpha,
    //     A, lda,  // leading dimension of A. If the matrix is ​​stored row-major, lda should be the number of columns in the matrix. Otherwise, lda should be the number of rows.
    //     B, ldb,  // leading dimension of B
    //     &beta,
    //     C, ldc   // C 的 leading dimension
    // );
    // printf("CHECK THE PARAM OF hipblasSgemm: %d %d %d %d %d %d\n", row_obj_constant, col_obj_constant, dims_, dims_, col_obj , row_obj_constant);
    // Strongly recommend reading this blog: https://blog.csdn.net/HaoBBNuanMM/article/details/103054357
    hipblasSgemm(cublas_handle_, HIPBLAS_OP_N, HIPBLAS_OP_N, size, 1,  dims_ + 1, &alpha, param_matrix, size, objective_matrix, col_obj, &beta, evaluate_score_, size);

    // printf("CHECK THE PARAM OF hipblasSgemm: %d %d %d %d %d %d\n", col_constraint, size, dims_+1, dims_+1, col_constraint , size);
    hipblasSgemm(cublas_handle_, HIPBLAS_OP_N, HIPBLAS_OP_N, size, row_constraint, dims_ + 1, &alpha, param_matrix, size, constraint_matrix, col_constraint, &beta, tmp_score, size);

    InequalityMask<<<1, size * row_constraint, 0, cuda_utils_->streams_[task_id_]>>>(tmp_score);

    hipblasSgemm(cublas_handle_, HIPBLAS_OP_N, HIPBLAS_OP_N, size, 1, col_lambda, &alpha, tmp_score, size, lambda_matrix, col_lambda, &beta, evaluate_score_, size);

    if(DEBUG_PRINT_FLAG || DEBUG_PRINT_EVALUATE_FLAG){
        // GPU DEBUG
        // printf("device obj_constant_matrix\n");
        // printMatrix<<<1, row_obj_constant*col_obj_constant, 0, cuda_utils_->streams_[task_id_]>>>(obj_constant_matrix);

        CHECK_CUDA(hipMemcpy(host_tmp_score, tmp_score, size * row_constraint * sizeof(float), hipMemcpyDeviceToHost));
        CHECK_CUDA(hipMemcpy(host_evaluate_score_, evaluate_score_, size * sizeof(float), hipMemcpyDeviceToHost));
        // CHECK_CUDA(hipMemcpy(host_param_matrix, param_matrix, (dims_ + 1) * size * sizeof(float), hipMemcpyDeviceToHost));
        CHECK_CUDA(hipStreamSynchronize(cuda_utils_->streams_[task_id_]));

        // check tmp score with individual after matrix multiplication
        PrintTmpScoreWithParam(host_tmp_score, host_param_matrix, row_constraint, size, dims_ + 1, "CHECK TMP SCORE");
        // check fitness with individual after matrix multiplication
        PrintFitnesssWithParam(host_evaluate_score_, host_param_matrix, 1, size, dims_ + 1, "CHECK FITNESS WITH PARAM");
    }
    UpdateFitnessBasedMatrix<64><<<1, size, 0, cuda_utils_->streams_[task_id_]>>>(new_cluster_data_, evaluate_score_);

    // (Abandoned) Use for loop to evaluate 
    // MainEvaluation<64><<<1, size, 0, cuda_utils_->streams_[task_id_]>>>(evolve_data_, new_cluster_data_);
}

void CudaDiffEvolveSolver::Evolution(int epoch, CudaEvolveType search_type){
    DuplicateBestAndReorganize<<<CUDA_PARAM_MAX_SIZE, 192, 0, cuda_utils_->streams_[task_id_]>>>(epoch, old_cluster_data_, 2);
    CudaEvolveProcess<<<CUDA_SOLVER_POP_SIZE, CUDA_PARAM_MAX_SIZE, 0, cuda_utils_->streams_[task_id_]>>>(epoch, old_cluster_data_, new_cluster_data_, random_center_->uniform_data_, random_center_->normal_data_, evolve_data_, CUDA_SOLVER_POP_SIZE, true);
    Evaluation(CUDA_SOLVER_POP_SIZE, epoch);

    // CHECK_CUDA(hipStreamSynchronize(cuda_utils_->streams_[task_id_]));
    UpdateParameter<64><<<CUDA_PARAM_MAX_SIZE, 128, 0, cuda_utils_->streams_[task_id_]>>>(epoch, evolve_data_, new_cluster_data_, old_cluster_data_, terminate_flag, last_fitness);

    CHECK_CUDA(hipMemcpyAsync(h_terminate_flag, terminate_flag, sizeof(int), hipMemcpyDeviceToHost, cuda_utils_->streams_[task_id_]));
    CHECK_CUDA(hipStreamSynchronize(cuda_utils_->streams_[task_id_]));
}

void CudaDiffEvolveSolver::InitSolver(int gpu_device, hipblasHandle_t handle, int task_id, CudaRandomCenter *random_center, Problem* host_problem, CudaParamIndividual *output_sol, const CudaVector<CudaParamIndividual, CUDA_MAX_POTENTIAL_SOLUTION> *last_potential_sol){
    if(DEBUG_ENABLE_NVTX)   init_range = nvtxRangeStart("Init Different Evolution Solver");

    gpu_device_ = gpu_device;
    random_center_ = random_center;

    CHECK_CUDA(hipSetDevice(gpu_device_));
    if (DEBUG_PRINT_FLAG || DEBUG_PRINT_INIT_SOLVER_FLAG) printf("CUDA SET DEVICE\n");

    con_var_dims_ = host_problem->num_continous;
    int_var_dims_ = host_problem->num_int;
    dims_ = host_problem->num_continous + host_problem->num_int;

    // initial constraint matrix
    row_constraint = host_problem->row_constraint_mat;
    col_constraint = host_problem->col_constraint_mat;
    
    row_obj = host_problem->row_objective_mat;
    col_obj = host_problem->col_objective_mat;

    row_lambda = host_problem->row_lambda;
    col_lambda = host_problem->col_lambda;

    task_id_ = task_id;

    // if(task_id == 0)   MallocSetup();
    MallocSetup();

    InitDiffEvolveParam();
    
    if (DEBUG_PRINT_FLAG || DEBUG_PRINT_INIT_SOLVER_FLAG) printf("INIT PARAM FOR DE\n");

    // Initial evolve data
    host_evolve_data_->problem_param.top_ratio = top_;
    // host_evolve_data_->new_cluster_vec = new_cluster_vec_;
    host_evolve_data_->problem_param.int_var_dims = int_var_dims_;
    host_evolve_data_->problem_param.con_var_dims = con_var_dims_;
    host_evolve_data_->problem_param.dims = int_var_dims_ + con_var_dims_;
    
    // // (Abandoned) Use for loop to evaluate 
    // // constraint
    // host_evolve_data_->num_constraint = host_problem->num_constraint;
    // host_evolve_data_->num_constraint_variable = host_problem->num_constraint_variable;
    // for (int i = 0; i < host_problem->num_constraint; ++i){
    //     for(int j = 0; j < host_problem->num_constraint_variable; ++j){
    //         host_evolve_data_->constraint_para[i][j] = host_problem->constraint_param[i][j];
    //     }
    // }

    // (Abandoned) Use for loop to evaluate 
    // // objective
    // host_evolve_data_->num_objective_param = host_problem->num_objective_param;
    // for (int i = 0; i < host_problem->num_objective_param; ++i){
    //     host_evolve_data_->objective_param[i] = host_problem->objective_param[i];
    // }

    size_t size_constraint_mat = row_constraint * col_constraint * sizeof(float);
    size_t size_obj = row_obj * col_obj * sizeof(float);

    host_evolve_data_->problem_param.max_lambda = host_problem->max_lambda;
    host_evolve_data_->problem_param.init_lambda = host_problem->init_lambda;
    host_evolve_data_->problem_param.max_round = host_problem->max_evolve_round;

    host_evolve_data_->problem_param.accuracy_rng = host_problem->accuracy_rng;
    host_evolve_data_->problem_param.elite_eval_count = host_problem->elite_eval_count;

    if (DEBUG_PRINT_FLAG || DEBUG_PRINT_INIT_SOLVER_FLAG) printf("start initialize cuBLAS handle\n");

    cublas_handle_ = handle;

    if (DEBUG_PRINT_FLAG || DEBUG_PRINT_INIT_SOLVER_FLAG) printf("finish initialize cuBLAS handle\n");
    
    *h_terminate_flag = 0;
    hipMemset(terminate_flag, 0, sizeof(int));
    float init_last_f = CUDA_MAX_FLOAT;
    CHECK_CUDA(hipMemcpy(last_fitness, &init_last_f, sizeof(float), hipMemcpyHostToDevice));

    if(DEBUG_ENABLE_NVTX)   setting_boundary_range = nvtxRangeStart("Init_Solver Setting Boundary");

    SetBoundary(host_problem);

    if (DEBUG_ENABLE_NVTX)  nvtxRangeEnd(setting_boundary_range);

    if(DEBUG_ENABLE_NVTX)   loading_last_sol_range = nvtxRangeStart("Init_Solver last solution");

    if (last_potential_sol != nullptr){
        for(int i = 0; i < last_potential_sol->len; ++i){
            memcpy(&last_potential_sol_.data[last_potential_sol_.len], &last_potential_sol->data[i], sizeof(CudaParamIndividual));
            for (int j = 0; j < dims_; j++) {
                last_potential_sol_.data[last_potential_sol_.len].param[j] = std::max(last_potential_sol_.data[last_potential_sol_.len].param[j], host_lower_bound_[j]);
                last_potential_sol_.data[last_potential_sol_.len].param[j] = std::min(last_potential_sol_.data[last_potential_sol_.len].param[j], host_upper_bound_[j]);
            }
            last_potential_sol_.len++;
        }
    }

    if (DEBUG_ENABLE_NVTX)  nvtxRangeEnd(loading_last_sol_range);

    host_evolve_data_->last_potential_sol = last_potential_sol_;
    
    if (DEBUG_PRINT_FLAG || DEBUG_PRINT_INIT_SOLVER_FLAG) printf("START MEMORY ASYNC\n");

    // Host --> GPU device
    // Split evolve_data_data content for asynchronous transmission
    // CHECK_CUDA(hipMemcpyAsync(&evolve_data_->problem_param, &host_evolve_data_->problem_param, sizeof(CudaProblemParam), hipMemcpyHostToDevice, cuda_utils_->streams_[task_id_]));
    // CHECK_CUDA(hipMemcpyAsync(evolve_data_->lower_bound, host_evolve_data_->lower_bound, CUDA_PARAM_MAX_SIZE * sizeof(float), hipMemcpyHostToDevice, cuda_utils_->streams_[1]));
    // CHECK_CUDA(hipMemcpyAsync(evolve_data_->upper_bound, host_evolve_data_->upper_bound, CUDA_PARAM_MAX_SIZE * sizeof(float), hipMemcpyHostToDevice, cuda_utils_->streams_[1]));
    // CHECK_CUDA(hipMemcpyAsync(&evolve_data_->hist_lshade_param, &host_evolve_data_->hist_lshade_param, sizeof(CudaLShadePair), hipMemcpyHostToDevice, cuda_utils_->streams_[1]));
    // CHECK_CUDA(hipMemcpyAsync(&evolve_data_->last_potential_sol, &host_evolve_data_->last_potential_sol, sizeof(CudaVector<CudaParamIndividual, CUDA_MAX_POTENTIAL_SOLUTION>), hipMemcpyHostToDevice, cuda_utils_->streams_[2]));
    
    CHECK_CUDA(hipMemcpyAsync(evolve_data_, host_evolve_data_, sizeof(CudaEvolveData), hipMemcpyHostToDevice, cuda_utils_->streams_[task_id_]));
    CHECK_CUDA(hipMemcpyAsync(constraint_matrix, host_problem->constraint_mat, size_constraint_mat, hipMemcpyHostToDevice, cuda_utils_->streams_[task_id_]));
    CHECK_CUDA(hipMemcpyAsync(objective_matrix, host_problem->objective_mat, size_obj, hipMemcpyHostToDevice, cuda_utils_->streams_[task_id_]));
    CHECK_CUDA(hipMemcpyAsync(lambda_matrix, host_problem->lambda_mat, row_lambda * col_lambda * sizeof(float), hipMemcpyHostToDevice, cuda_utils_->streams_[task_id_])); 

    CHECK_CUDA(hipStreamSynchronize(cuda_utils_->streams_[task_id_]));
    // CHECK_CUDA(hipStreamSynchronize(cuda_utils_->streams_[1]));
    // CHECK_CUDA(hipStreamSynchronize(cuda_utils_->streams_[2]));

    // if (last_sol == nullptr){
    //     CHECK_CUDA(hipStreamSynchronize(cuda_utils_->streams_[task_id_]));
    // }

    if (DEBUG_PRINT_FLAG || DEBUG_PRINT_INIT_SOLVER_FLAG) printf("MEMORY ASYNC SUBMIT\n");

    InitCudaEvolveData<<<1, CUDA_SOLVER_POP_SIZE, 0, cuda_utils_->streams_[task_id_]>>>(evolve_data_, old_cluster_data_, CUDA_SOLVER_POP_SIZE);


    WarmStart(host_problem, output_sol);

    // if (DEBUG_PRINT_FLAG){
    //     CHECK_CUDA(hipStreamSynchronize(cuda_utils_->streams_[task_id_]));
    //     CHECK_CUDA(hipMemcpyAsync(host_new_cluster_data_, new_cluster_data_, sizeof(CudaParamClusterData<64>), hipMemcpyDeviceToHost, cuda_utils_->streams_[task_id_]));
    //     PrintClusterData(host_new_cluster_data_);
    //     // CHECK_CUDA(hipMemcpyAsync(host_old_cluster_data_, old_cluster_data_, sizeof(CudaParamClusterData<192>), hipMemcpyDeviceToHost, cuda_utils_->streams_[task_id_]));
    //     // PrintClusterData(host_old_cluster_data_);

    //     // CHECK_CUDA(hipMemcpyAsync(host_evolve_data_, evolve_data_, sizeof(CudaEvolveData), hipMemcpyDeviceToHost, cuda_utils_->streams_[task_id_]));
    //     // printf("CUDA_MAX_FLOAT %f\n", CUDA_MAX_FLOAT);
    // }
}

__global__ void LoadWarmStartResultForSolver(CudaEvolveData *evolve, CudaParamClusterData<64> *new_param){
    ConvertCudaParam<64>(new_param, &evolve->warm_start, blockIdx.x, threadIdx.x);
}

template <int T=192>
__global__ void GetSolFromOldParam(CudaParamClusterData<192> *old_param, CudaParamIndividual *solution){
    ConvertCudaParamRevert<192>(old_param, solution, blockIdx.x, threadIdx.x);
}

CudaParamIndividual CudaDiffEvolveSolver::Solver(){
    // nvtx3::mark("Different Evolvution Solver!");
    if(DEBUG_ENABLE_NVTX)   solver_range = nvtxRangeStart("Different Evolvution Solver");

    init_pop_size_ = CUDA_SOLVER_POP_SIZE;
    pop_size_ = CUDA_SOLVER_POP_SIZE;

    InitCudaEvolveData<<<1, CUDA_SOLVER_POP_SIZE, 0, cuda_utils_->streams_[task_id_]>>>(evolve_data_, old_cluster_data_, CUDA_SOLVER_POP_SIZE);

    InitParameter<<<1, CUDA_SOLVER_POP_SIZE, 0, cuda_utils_->streams_[task_id_]>>>(evolve_data_, CUDA_SOLVER_POP_SIZE, new_cluster_data_, old_cluster_data_, random_center_->uniform_data_);

    LoadWarmStartResultForSolver<<<1, CUDA_PARAM_MAX_SIZE, 0, cuda_utils_->streams_[task_id_]>>>(evolve_data_, new_cluster_data_);

    // based on warm start result to generate 
    GenerativeRandSolNearBest<<<CUDA_SOLVER_POP_SIZE, CUDA_PARAM_MAX_SIZE, 0, cuda_utils_->streams_[task_id_]>>>(evolve_data_, new_cluster_data_, random_center_->uniform_data_, 16, 0.1, 0.1, CUDA_SOLVER_POP_SIZE);

    Evaluation(CUDA_SOLVER_POP_SIZE, 0);

    ParaFindMax2<CUDA_SOLVER_POP_SIZE, 64><<<1, CUDA_SOLVER_POP_SIZE, 0, cuda_utils_->streams_[task_id_]>>>(new_cluster_data_);

    SaveNewParamAsOldParam<<<CUDA_SOLVER_POP_SIZE, CUDA_PARAM_MAX_SIZE, 0, cuda_utils_->streams_[task_id_]>>>(new_cluster_data_, old_cluster_data_, 0, CUDA_SOLVER_POP_SIZE, 0);

    for (int i = 0; i < host_evolve_data_->problem_param.max_round && !*h_terminate_flag; ++i) {
        // printf("generation i:%d\n", i);
        Evolution(i, CudaEvolveType::GLOBAL);
    }

    if (DEBUG_PRINT_FLAG || DEBUG_PRINT_SOLVER_FLAG){
        // CHECK_CUDA(hipMemcpyAsync(host_new_cluster_data_, new_cluster_data_, sizeof(CudaParamClusterData<64>), hipMemcpyDeviceToHost, cuda_utils_->streams_[task_id_]));
        // PrintClusterData(host_new_cluster_data_);
        CHECK_CUDA(hipMemcpyAsync(host_old_cluster_data_, old_cluster_data_, sizeof(CudaParamClusterData<192>), hipMemcpyDeviceToHost, cuda_utils_->streams_[task_id_]));
        CHECK_CUDA(hipStreamSynchronize(cuda_utils_->streams_[task_id_]));
        PrintClusterData<192>(host_old_cluster_data_);

        // CHECK_CUDA(hipMemcpyAsync(host_evolve_data_, evolve_data_, sizeof(CudaEvolveData), hipMemcpyDeviceToHost, cuda_utils_->streams_[task_id_]));
        // printf("CUDA_MAX_FLOAT %f\n", CUDA_MAX_FLOAT);
    }
    
    // Get the first individual from old param (after sorting, the first one is the best one)
    GetSolFromOldParam<192><<<1, CUDA_PARAM_MAX_SIZE, 0, cuda_utils_->streams_[task_id_]>>>(old_cluster_data_, result);
    CHECK_CUDA(hipMemcpyAsync(host_result, result, sizeof(CudaParamIndividual), hipMemcpyDeviceToHost, cuda_utils_->streams_[task_id_]));
    CHECK_CUDA(hipStreamSynchronize(cuda_utils_->streams_[task_id_]));

    for(int i = con_var_dims_; i < dims_; ++i){
        host_result->param[i] = floor(host_result->param[i]);
    }

    // if (DEBUG_PRINT_FLAG || DEBUG_PRINT_SOLVER_FLAG)   printFinalResult(host_result->fitness, host_result->param, dims_);
    printFinalResult(host_result->fitness, host_result->param, dims_);

    if(DEBUG_ENABLE_NVTX)   nvtxRangeEnd(solver_range);

    CHECK_CUDA(hipDeviceSynchronize());

    return *host_result;
}

CudaDiffEvolveSolver::~CudaDiffEvolveSolver(){
    if (cudamalloc_flag){
        // GPU device
        CHECK_CUDA(hipFree(evolve_data_));
        CHECK_CUDA(hipFree(new_cluster_data_));
        CHECK_CUDA(hipFree(old_cluster_data_));
        // CHECK_CUDA(hipFree(new_cluster_vec_));
        CHECK_CUDA(hipFree(constraint_matrix));
        CHECK_CUDA(hipFree(objective_matrix));
        CHECK_CUDA(hipFree(param_matrix));
        CHECK_CUDA(hipFree(evaluate_score_));
        CHECK_CUDA(hipFree(tmp_score));
        CHECK_CUDA(hipFree(lambda_matrix));
        CHECK_CUDA(hipFree(result));

        // CPU host
        if (DEBUG_PRINT_FLAG || DEBUG_PRINT_SOLVER_FLAG){
            CHECK_CUDA(hipHostFree(host_new_cluster_data_));
            CHECK_CUDA(hipHostFree(host_old_cluster_data_));
        }

        if (DEBUG_PRINT_FLAG || DEBUG_PRINT_EVALUATE_FLAG){
            CHECK_CUDA(hipHostFree(host_evaluate_score_));
            CHECK_CUDA(hipHostFree(host_param_matrix));
            CHECK_CUDA(hipHostFree(host_tmp_score));
            CHECK_CUDA(hipHostFree(h_lambda_matrix));
            CHECK_CUDA(hipHostFree(h_constraint_matrix));
            CHECK_CUDA(hipHostFree(h_objective_matrix));
        }
        
        CHECK_CUDA(hipHostFree(host_evolve_data_));
        CHECK_CUDA(hipHostFree(host_result));
        
    }
    
}

}