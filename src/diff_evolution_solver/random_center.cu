#include "hip/hip_runtime.h"
#include "diff_evolution_solver/random_center.cuh"

namespace cudaprocess{
    __global__ void RndInit(hiprandState_t *states, long long unsigned seed, int total_size) {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idx < total_size) {
            hiprand_init(seed, idx, idx, &states[idx]);
        }
    }

    __global__ void GenUniformRandom(hiprandState_t *states, float *res, int total_size) {
        int idx = (blockIdx.x * blockDim.x + threadIdx.x) * 10;
        int state_idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idx + 9 < total_size) {
            hiprandState_t localState = states[state_idx];  // 本地副本提高效率
        #pragma unroll
            for (int i = 0; i < 10; ++i) {
            res[idx + i] = hiprand_uniform(&localState);
            if (res[idx + i] >= 1.0) {
                res[idx + i] = 0.;
            }
            }
        }
    }

    __global__ void GenNormalRandom(hiprandState_t *states, float *res, int total_size) {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idx < total_size) {
            res[idx] = hiprand_normal(&states[idx]);
        }
    }

    // 添加一个辅助kernel来检查states的初始化
    __global__ void CheckStates(hiprandState_t *states, int n) {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idx < n) {
            // 生成一个测试随机数
            float test = hiprand_uniform(&states[idx]);
            printf("State[%d] test random: %f\n", idx, test);
        }
    }

    CudaRandomCenter::CudaRandomCenter(int gpu_device)
    {
        normal_size_ = size_ * 3 * CUDA_MAX_ROUND_NUM;
        uniform_size_ = 512000;
        int state_size_ = 51200;
        CHECK_CUDA(hipSetDevice(gpu_device));
        CHECK_CUDA(hipMalloc(&states_, sizeof(hiprandState_t) * state_size_));
        CHECK_CUDA(hipMalloc(&uniform_data_, sizeof(float) * uniform_size_));
        CHECK_CUDA(hipMalloc(&normal_data_, sizeof(float) * normal_size_));
        // auto seed_ = 0;
        // 使用时间作为种子
        auto seed_ = static_cast<long long unsigned>(time(nullptr));
        RndInit<<<50, 1024>>>(states_, seed_, state_size_);

        normal_grid_size_ = (normal_size_ - 1) / 1024 + 1;
        // uniform_grid_size_ = (uniform_size_ - 1) / 1024 + 1;
        uniform_grid_size_ = 50;
        Generate();
        hipDeviceSynchronize();

        printf("\nChecking random states:\n");
        CheckStates<<<50, 1024>>>(states_, state_size_);
        hipDeviceSynchronize();
    }
    
    CudaRandomCenter::~CudaRandomCenter()
    {
        CHECK_CUDA(hipFree(states_));
        CHECK_CUDA(hipFree(uniform_data_));
        CHECK_CUDA(hipFree(normal_data_));
    }

    void CudaRandomCenter::Generate() {
        GenUniformRandom<<<uniform_grid_size_, 1024>>>(states_, uniform_data_, uniform_size_);
        GenNormalRandom<<<normal_grid_size_, 1024>>>(states_, normal_data_, normal_size_);
        hipDeviceSynchronize();
    }

    __device__ __forceinline__ float UniformReal(hiprandState_t *states, float l, float r) { return l + hiprand_uniform(states) * (r - l); }

    __device__ __forceinline__ float NormalReal(hiprandState_t *states, float miu, float sigma) { return hiprand_normal(states) * sigma + miu; }

    __device__ __forceinline__ int UniformInt(hiprandState_t *states, int l, int r) {
        int tmp = l + hiprand_uniform(states) * (r + 1 - l);
        if (tmp == r + 1) {
            tmp = r;
        }
        return tmp;
    }
}